#include "hip/hip_runtime.h"
#include <iostream>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/sort.h>
#include <thrust/generate.h>
#include <thrust/copy.h>
#include <algorithm>
#include <cstdlib>


using namespace std;


#include <chrono>
#include <stack>
std::stack<clock_t> tictoc_stack;
void tic() {
	tictoc_stack.push(clock());
}

void toc() {
	std::cout << "Time elapsed: "
		<< ((double)(clock() - tictoc_stack.top())) / CLOCKS_PER_SEC
		<< std::endl;
	tictoc_stack.pop();
}


void thrustSort(float *V, int *K, int N)
{
	thrust::device_vector<float> d_V(V, V+N);
	thrust::device_vector<int> d_K(N);
	thrust::sequence(d_K.begin(), d_K.end(), 0, 1);
	thrust::sort_by_key(d_V.begin(), d_V.end(), d_K.begin());
	thrust::copy(d_K.begin(), d_K.end(), K);
	thrust::copy(d_V.begin(), d_V.end(), V);
}

int main() {
	hipFree(0);
	int size = 100*1024*1024;
	thrust::host_vector<float> h_values(size);
	thrust::host_vector<int> h_keys(size);
	
	std::generate(h_values.begin(), h_values.end(), rand);

	thrustSort(h_values.data(), h_keys.data(), h_values.size());

	if (size < 100)
	{
		for (int i = 0; i < size; i++)
		{
			cout << "(" << h_values[i] << ", " << h_keys[i] << ")";
		}
	}

	return 0;
}