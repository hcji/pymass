#include "hip/hip_runtime.h"
#include <iostream>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/sort.h>
#include <thrust/generate.h>
#include <thrust/copy.h>
#include <algorithm>
#include <cstdlib>
#include "cuda_sort.h"


using namespace std;



std::stack<clock_t> gtictoc_stack;
void gtic() {
	gtictoc_stack.push(clock());
}

void gtoc() {
	std::cout << "Time elapsed: "
		<< ((double)(clock() - gtictoc_stack.top())) / CLOCKS_PER_SEC
		<< std::endl;
	gtictoc_stack.pop();
}


void thrustSort(float *V, int *K, int N)
{
	thrust::device_vector<float> d_V(V, V+N);
	thrust::device_vector<int> d_K(N);
	thrust::sequence(d_K.begin(), d_K.end(), 0, 1);
	thrust::sort_by_key(d_V.begin(), d_V.end(), d_K.begin(), thrust::greater<float>());
	thrust::copy(d_K.begin(), d_K.end(), K);
	thrust::copy(d_V.begin(), d_V.end(), V);
}

void processLCMS(LCMS & lcms)
{
	hipFree(0);
	cout << "using lcms object in CUDA, and its scan size is: " << lcms.m_massScans.size() << endl;

	Eigen::MatrixXf rmv = lcms.getAll();
	Eigen::VectorXi ids(rmv.rows());
	gtic();
	thrustSort(rmv.col(2).data(), ids.data(), rmv.rows());
	gtoc();

	//cout << lcms.m_massScans[0].mz << endl;
	cout << rmv.col(2).head(20) << endl;
	cout << "##########################"<<endl;
	cout << ids.head(20) << endl;
}

