#include "hip/hip_runtime.h"
#include <iostream>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/sort.h>
#include <thrust/generate.h>
#include <thrust/copy.h>
#include <algorithm>
#include <cstdlib>
#include "cuda_sort.h"


using namespace std;



std::stack<clock_t> gtictoc_stack;
void gtic() {
	gtictoc_stack.push(clock());
}

void gtoc() {
	std::cout << "Time elapsed: "
		<< ((double)(clock() - gtictoc_stack.top())) / CLOCKS_PER_SEC
		<< std::endl;
	gtictoc_stack.pop();
}

void sort_by_col(Eigen::MatrixXf & m, int col)
{
	for (int i = 0; i < m.cols(); i++)
	{
		thrust::device_vector<float> d_K(m.col(col).data(), m.col(col).data() + m.rows());
		thrust::device_vector<float> d_V(m.col(i).data(), m.col(i).data() + m.rows());
		thrust::sort_by_key(d_K.begin(), d_K.end(), d_V.begin(), thrust::greater<float>());
		if (i!=col)
		{
			thrust::copy(d_V.begin(), d_V.end(), m.col(i).data());
		}
		if (i == m.cols()-1)
		{
			thrust::copy(d_K.begin(), d_K.end(), m.col(col).data());
		}
	}
}

std::set<Eigen::VectorXf> pic_seed(const Eigen::MatrixXf & m, float mz_tol)
{
	std::set<Eigen::VectorXf> ret;
	return ret;
}

void processLCMS(LCMS & lcms)
{
	hipFree(0);
	cout << "using lcms object in CUDA, and its scan size is: " << lcms.m_massScans.size() << endl;

	Eigen::MatrixXf rmv = lcms.getAll();

	gtic();
	sort_by_col(rmv, 2);
	gtoc();

	gtic();
	pic_seed(rmv, 0.5);
	gtoc();

	cout << rmv.topRows(10) << endl;
	cout << "##########################"<<endl;
}

