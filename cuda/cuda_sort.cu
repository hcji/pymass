#include "hip/hip_runtime.h"
#include <iostream>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/sort.h>
#include <thrust/generate.h>
#include <thrust/copy.h>
#include <algorithm>
#include <cstdlib>
#include "cuda_sort.h"


using namespace std;



namespace Kernel
{

	static void HandleError(hipError_t err, const char *file, int line)
	{
		if (err != hipSuccess)
		{
			printf("%s in %s at line %d\n", hipGetErrorString(err), file, line);
			exit(EXIT_FAILURE);
		}
	}

    #define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))

	__global__ void finc_pics_k(Eigen::Vector3f *v1, Eigen::Vector3f *v2, float *out, size_t N)
	{
		int idx = blockIdx.x * blockDim.x + threadIdx.x;
		if (idx < N)
		{
			out[idx] = v1[idx].dot(v2[idx]);
		}
		return;
	}
	
	double find_pics(const std::vector<Eigen::Vector3f> & v1, const std::vector<Eigen::Vector3f> & v2)
	{
		int n = v1.size();
		float *ret = new float[n];

		Eigen::Vector3f *dev_v1, *dev_v2;
		HANDLE_ERROR(hipMalloc((void **)&dev_v1, sizeof(Eigen::Vector3f)*n));
		HANDLE_ERROR(hipMalloc((void **)&dev_v2, sizeof(Eigen::Vector3f)*n));
		float* dev_ret;
		HANDLE_ERROR(hipMalloc((void **)&dev_ret, sizeof(float)*n));

		HANDLE_ERROR(hipMemcpy(dev_v1, v1.data(), sizeof(Eigen::Vector3f)*n, hipMemcpyHostToDevice));
		HANDLE_ERROR(hipMemcpy(dev_v2, v2.data(), sizeof(Eigen::Vector3f)*n, hipMemcpyHostToDevice));

		finc_pics_k << <(n + 1023) / 1024, 1024 >> > (dev_v1, dev_v2, dev_ret, n);

		HANDLE_ERROR(hipMemcpy(ret, dev_ret, sizeof(float)*n, hipMemcpyDeviceToHost));

		for (int i = 1; i < n; ++i)
		{
			ret[0] += ret[i];
		}

		hipFree(dev_v1);
		hipFree(dev_v2);
		hipFree(dev_ret);

		return ret[0];
	}
}



std::stack<clock_t> gtictoc_stack;
void gtic() {
	gtictoc_stack.push(clock());
}

void gtoc() {
	std::cout << "Time elapsed: "
		<< ((double)(clock() - gtictoc_stack.top())) / CLOCKS_PER_SEC
		<< std::endl;
	gtictoc_stack.pop();
}

void printVV(const std::vector<Eigen::Vector3f> & vec, int n)
{
	Eigen::MatrixXf m(vec.size(), 3);
	int i = 0;
	std::for_each(vec.begin(), vec.end(), [&m, &i](const Eigen::Vector3f & v) {
		m.row(i) = v;
		i++; });
	if (n<= vec.size() && n>0)
	{
		cout << m.topRows(n) << endl;
	}
	else
	{
		cout << m << endl;
	}
}

void sort_by_col(Eigen::MatrixXf & m, int col)
{
	for (int i = 0; i < m.cols(); i++)
	{
		thrust::device_vector<float> d_K(m.col(col).data(), m.col(col).data() + m.rows());
		thrust::device_vector<float> d_V(m.col(i).data(), m.col(i).data() + m.rows());
		thrust::sort_by_key(d_K.begin(), d_K.end(), d_V.begin(), thrust::greater<float>());
		if (i!=col)
		{
			thrust::copy(d_V.begin(), d_V.end(), m.col(i).data());
		}
		if (i == m.cols()-1)
		{
			thrust::copy(d_K.begin(), d_K.end(), m.col(col).data());
		}
	}
}


std::vector<Eigen::Vector3f> pic_seeds(const Eigen::MatrixXf & m, float mz_tol, int num_seed)
{
	auto comp = [](const Eigen::VectorXf& lhs, const Eigen::VectorXf& rhs) -> bool {
		return lhs[1] < rhs[1]; 
	};
	std::set<Eigen::VectorXf, mz_comp> seed_set(comp);

	for (int i =0; i< m.rows(); i++)
	{
		auto it = seed_set.lower_bound(m.row(i));
		if (seed_set.size()==0)
		{
			seed_set.insert(m.row(i));
		}
		else
		{
			if (it == seed_set.end())
			{
				if (m.row(i)[1] - (*std::prev(seed_set.end()))[1] > mz_tol)
				{
					seed_set.insert(m.row(i));
				}
			}
			else if (it == seed_set.begin())
			{
				if ((*seed_set.begin())[1] - m.row(i)[1] > mz_tol)
				{
					seed_set.insert(m.row(i));
				}
			}
			else
			{
				if (  ((*it)[1] - m.row(i)[1] > mz_tol ) && 
					  ( m.row(i)[1] - (*std::prev(it))[1]> mz_tol))
				{
					seed_set.insert(m.row(i));
				}
			}
		}

		if (seed_set.size()==num_seed)
		{
			break;
		}
	}

	std::vector<Eigen::Vector3f> ret(seed_set.size());

	int i = 0;
	std::for_each(seed_set.begin(), seed_set.end(), [&ret, &i](const Eigen::VectorXf & v) {
		ret[i] = v;
		i++; });

	return ret;
}

std::vector<std::vector<Eigen::Vector3f>> regions_of_seeds(LCMS & lcms, const std::vector<Eigen::Vector3f> & seeds, float peak_width, float mz_tol)
{
	std::vector<std::vector<Eigen::Vector3f>> regions;
	for (auto seed: seeds)
	{
		std::vector<Eigen::Vector3f> region = lcms.getRegion(seed[0] - peak_width, seed[0] + peak_width, seed[1] - mz_tol, seed[1] + mz_tol);
		regions.push_back(region);
	}
	return regions;
}

void processLCMS(LCMS & lcms)
{
	hipFree(0);
	cout << "using lcms object in NVCC, and its scan size is: " << lcms.m_massScans.size() << endl;

	Eigen::MatrixXf rmv = lcms.getAll();

	gtic();
	sort_by_col(rmv, 2);
	gtoc();

	gtic();
	std::vector<Eigen::Vector3f> seeds = pic_seeds(rmv, 0.05f, 4000);
	gtoc();

	gtic();
	std::vector<std::vector<Eigen::Vector3f>> regions = regions_of_seeds(lcms, seeds, 50.0f, 0.05f);
	gtoc();

	gtic();
	double x = Kernel::find_pics(seeds, seeds);
	gtoc();
	cout << "Calculated by CUDA kernel: " << x << endl;

	cout << "##########################"<<endl;
}