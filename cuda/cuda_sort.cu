#include "hip/hip_runtime.h"
#include <iostream>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/sort.h>
#include <thrust/generate.h>
#include <thrust/copy.h>
#include <algorithm>
#include <cstdlib>
#include "cuda_sort.h"


using namespace std;



namespace Kernel
{

	static void HandleError(hipError_t err, const char *file, int line)
	{
		if (err != hipSuccess)
		{
			printf("%s in %s at line %d\n", hipGetErrorString(err), file, line);
			exit(EXIT_FAILURE);
		}
	}

    #define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))

	__global__ void cu_dot(Eigen::Vector3d *v1, Eigen::Vector3d *v2, double *out, size_t N)
	{
		int idx = blockIdx.x * blockDim.x + threadIdx.x;
		if (idx < N)
		{
			out[idx] = v1[idx].dot(v2[idx]);
		}
		return;
	}
	
	double dot(const std::vector<Eigen::Vector3d> & v1, const std::vector<Eigen::Vector3d> & v2)
	{
		int n = v1.size();
		double *ret = new double[n];

		Eigen::Vector3d *dev_v1, *dev_v2;
		HANDLE_ERROR(hipMalloc((void **)&dev_v1, sizeof(Eigen::Vector3d)*n));
		HANDLE_ERROR(hipMalloc((void **)&dev_v2, sizeof(Eigen::Vector3d)*n));
		double* dev_ret;
		HANDLE_ERROR(hipMalloc((void **)&dev_ret, sizeof(double)*n));

		HANDLE_ERROR(hipMemcpy(dev_v1, v1.data(), sizeof(Eigen::Vector3d)*n, hipMemcpyHostToDevice));
		HANDLE_ERROR(hipMemcpy(dev_v2, v2.data(), sizeof(Eigen::Vector3d)*n, hipMemcpyHostToDevice));

		cu_dot << <(n + 1023) / 1024, 1024 >> > (dev_v1, dev_v2, dev_ret, n);

		HANDLE_ERROR(hipMemcpy(ret, dev_ret, sizeof(double)*n, hipMemcpyDeviceToHost));

		for (int i = 1; i < n; ++i)
		{
			ret[0] += ret[i];
		}

		return ret[0];
	}
}



std::stack<clock_t> gtictoc_stack;
void gtic() {
	gtictoc_stack.push(clock());
}

void gtoc() {
	std::cout << "Time elapsed: "
		<< ((double)(clock() - gtictoc_stack.top())) / CLOCKS_PER_SEC
		<< std::endl;
	gtictoc_stack.pop();
}

void sort_by_col(Eigen::MatrixXf & m, int col)
{
	for (int i = 0; i < m.cols(); i++)
	{
		thrust::device_vector<float> d_K(m.col(col).data(), m.col(col).data() + m.rows());
		thrust::device_vector<float> d_V(m.col(i).data(), m.col(i).data() + m.rows());
		thrust::sort_by_key(d_K.begin(), d_K.end(), d_V.begin(), thrust::greater<float>());
		if (i!=col)
		{
			thrust::copy(d_V.begin(), d_V.end(), m.col(i).data());
		}
		if (i == m.cols()-1)
		{
			thrust::copy(d_K.begin(), d_K.end(), m.col(col).data());
		}
	}
}


std::set<Eigen::VectorXf, mz_comp> pic_seed(const Eigen::MatrixXf & m, float mz_tol, int num_seed)
{
	auto comp = [](const Eigen::VectorXf& lhs, const Eigen::VectorXf& rhs) -> bool {
		return lhs[1] < rhs[1]; 
	};
	std::set<Eigen::VectorXf, mz_comp> ret(comp);

	for (int i =0; i< m.rows(); i++)
	{
		auto it = ret.lower_bound(m.row(i));
		if (ret.size()==0)
		{
			ret.insert(m.row(i));
		}
		else
		{
			if (it == ret.end())
			{
				if (m.row(i)[1] - (*std::prev(ret.end()))[1] > mz_tol)
				{
					ret.insert(m.row(i));
				}
			}
			else if (it == ret.begin())
			{
				if ((*ret.begin())[1] - m.row(i)[1] > mz_tol)
				{
					ret.insert(m.row(i));
				}
			}
			else
			{
				if (  ((*it)[1] - m.row(i)[1] > mz_tol ) && 
					  ( m.row(i)[1] - (*std::prev(it))[1]> mz_tol))
				{
					ret.insert(m.row(i));
				}
			}
		}

		if (ret.size()==num_seed)
		{
			break;
		}
	}


	Eigen::MatrixXf seed(ret.size(), 3);
	int i = 0;
	std::for_each(ret.begin(), ret.end(), [&seed, &i](const Eigen::VectorXf & v) {
		seed.row(i) = v;
		i++; });

	cout << seed << endl;

	return ret;
}

void processLCMS(LCMS & lcms)
{
	hipFree(0);
	cout << "using lcms object in CUDA, and its scan size is: " << lcms.m_massScans.size() << endl;

	Eigen::MatrixXf rmv = lcms.getAll();

	gtic();
	sort_by_col(rmv, 2);
	gtoc();

	gtic();
	pic_seed(rmv, 0.05, 50);
	gtoc();

	//cout << rmv.topRows(10) << endl;


	std::vector<Eigen::Vector3d> v1(4000, Eigen::Vector3d{ 1.0, 1.0, 1.0 });
	std::vector<Eigen::Vector3d> v2(4000, Eigen::Vector3d{ -1.0, 1.0, 1.0 });
	gtic();
	double x = Kernel::dot(v1, v2);
	gtoc();
	cout << "Dot calculated by CUDA kernel: " << x << endl;

	cout << "##########################"<<endl;
}