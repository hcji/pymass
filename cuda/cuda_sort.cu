#include "hip/hip_runtime.h"
#include <iostream>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/sort.h>
#include <thrust/generate.h>
#include <thrust/copy.h>
#include <algorithm>
#include <cstdlib>
#include "cuda_sort.h"


using namespace std;



std::stack<clock_t> gtictoc_stack;
void gtic() {
	gtictoc_stack.push(clock());
}

void gtoc() {
	std::cout << "Time elapsed: "
		<< ((double)(clock() - gtictoc_stack.top())) / CLOCKS_PER_SEC
		<< std::endl;
	gtictoc_stack.pop();
}


void thrustSort(float *V, int *K, int N)
{
	thrust::device_vector<float> d_V(V, V+N);
	thrust::device_vector<int> d_K(N);
	thrust::sequence(d_K.begin(), d_K.end(), 0, 1);
	thrust::sort_by_key(d_V.begin(), d_V.end(), d_K.begin());
	thrust::copy(d_K.begin(), d_K.end(), K);
	thrust::copy(d_V.begin(), d_V.end(), V);
}