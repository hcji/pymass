#include "hip/hip_runtime.h"
#include <iostream>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/sort.h>
#include <thrust/generate.h>
#include <thrust/copy.h>
#include <algorithm>
#include <cstdlib>
#include "cuda_sort.h"


using namespace std;



namespace Kernel
{

	static void HandleError(hipError_t err, const char *file, int line)
	{
		if (err != hipSuccess)
		{
			printf("%s in %s at line %d\n", hipGetErrorString(err), file, line);
			exit(EXIT_FAILURE);
		}
	}

    #define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))

	__global__ void cu_dot(Eigen::Vector3f *v1, Eigen::Vector3f *v2, float *out, size_t N)
	{
		int idx = blockIdx.x * blockDim.x + threadIdx.x;
		if (idx < N)
		{
			out[idx] = v1[idx].dot(v2[idx]);
		}
		return;
	}
	
	double dot(const std::vector<Eigen::Vector3f> & v1, const std::vector<Eigen::Vector3f> & v2)
	{
		int n = v1.size();
		float *ret = new float[n];

		Eigen::Vector3f *dev_v1, *dev_v2;
		HANDLE_ERROR(hipMalloc((void **)&dev_v1, sizeof(Eigen::Vector3f)*n));
		HANDLE_ERROR(hipMalloc((void **)&dev_v2, sizeof(Eigen::Vector3f)*n));
		float* dev_ret;
		HANDLE_ERROR(hipMalloc((void **)&dev_ret, sizeof(float)*n));

		HANDLE_ERROR(hipMemcpy(dev_v1, v1.data(), sizeof(Eigen::Vector3f)*n, hipMemcpyHostToDevice));
		HANDLE_ERROR(hipMemcpy(dev_v2, v2.data(), sizeof(Eigen::Vector3f)*n, hipMemcpyHostToDevice));

		cu_dot << <(n + 1023) / 1024, 1024 >> > (dev_v1, dev_v2, dev_ret, n);

		HANDLE_ERROR(hipMemcpy(ret, dev_ret, sizeof(float)*n, hipMemcpyDeviceToHost));

		for (int i = 1; i < n; ++i)
		{
			ret[0] += ret[i];
		}

		return ret[0];
	}
}



std::stack<clock_t> gtictoc_stack;
void gtic() {
	gtictoc_stack.push(clock());
}

void gtoc() {
	std::cout << "Time elapsed: "
		<< ((double)(clock() - gtictoc_stack.top())) / CLOCKS_PER_SEC
		<< std::endl;
	gtictoc_stack.pop();
}

void sort_by_col(Eigen::MatrixXf & m, int col)
{
	for (int i = 0; i < m.cols(); i++)
	{
		thrust::device_vector<float> d_K(m.col(col).data(), m.col(col).data() + m.rows());
		thrust::device_vector<float> d_V(m.col(i).data(), m.col(i).data() + m.rows());
		thrust::sort_by_key(d_K.begin(), d_K.end(), d_V.begin(), thrust::greater<float>());
		if (i!=col)
		{
			thrust::copy(d_V.begin(), d_V.end(), m.col(i).data());
		}
		if (i == m.cols()-1)
		{
			thrust::copy(d_K.begin(), d_K.end(), m.col(col).data());
		}
	}
}


std::set<Eigen::VectorXf, mz_comp> pic_seed(const Eigen::MatrixXf & m, float mz_tol, int num_seed)
{
	auto comp = [](const Eigen::VectorXf& lhs, const Eigen::VectorXf& rhs) -> bool {
		return lhs[1] < rhs[1]; 
	};
	std::set<Eigen::VectorXf, mz_comp> ret(comp);

	for (int i =0; i< m.rows(); i++)
	{
		auto it = ret.lower_bound(m.row(i));
		if (ret.size()==0)
		{
			ret.insert(m.row(i));
		}
		else
		{
			if (it == ret.end())
			{
				if (m.row(i)[1] - (*std::prev(ret.end()))[1] > mz_tol)
				{
					ret.insert(m.row(i));
				}
			}
			else if (it == ret.begin())
			{
				if ((*ret.begin())[1] - m.row(i)[1] > mz_tol)
				{
					ret.insert(m.row(i));
				}
			}
			else
			{
				if (  ((*it)[1] - m.row(i)[1] > mz_tol ) && 
					  ( m.row(i)[1] - (*std::prev(it))[1]> mz_tol))
				{
					ret.insert(m.row(i));
				}
			}
		}

		if (ret.size()==num_seed)
		{
			break;
		}
	}

	return ret;
}

void processLCMS(LCMS & lcms)
{
	hipFree(0);
	cout << "using lcms object in CUDA, and its scan size is: " << lcms.m_massScans.size() << endl;

	Eigen::MatrixXf rmv = lcms.getAll();

	gtic();
	sort_by_col(rmv, 2);
	gtoc();

	gtic();
	std::set<Eigen::VectorXf, mz_comp> pic_seed_set = pic_seed(rmv, 0.05f, 4000);
	gtoc();

	std::vector<Eigen::Vector3f> v1(pic_seed_set.size());
	std::vector<Eigen::Vector3f> v2(pic_seed_set.size());

	int i = 0;
	std::for_each(pic_seed_set.begin(), pic_seed_set.end(), [&v1, &v2, &i](const Eigen::VectorXf & v) {
		v1[i] = v; v2[i] = v;
		i++; });


	gtic();
	double x = Kernel::dot(v1, v2);
	gtoc();
	cout << "Dot calculated by CUDA kernel: " << x << endl;

	cout << "##########################"<<endl;
}