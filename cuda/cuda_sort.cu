#include "hip/hip_runtime.h"
#include <iostream>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/sort.h>
#include <thrust/generate.h>
#include <thrust/copy.h>
#include <algorithm>
#include <cstdlib>
#include "cuda_sort.h"


using namespace std;



std::stack<clock_t> gtictoc_stack;
void gtic() {
	gtictoc_stack.push(clock());
}

void gtoc() {
	std::cout << "Time elapsed: "
		<< ((double)(clock() - gtictoc_stack.top())) / CLOCKS_PER_SEC
		<< std::endl;
	gtictoc_stack.pop();
}

void sort_by_col(Eigen::MatrixXf & m, int col)
{
	for (int i = 0; i < m.cols(); i++)
	{
		thrust::device_vector<float> d_K(m.col(col).data(), m.col(col).data() + m.rows());
		thrust::device_vector<float> d_V(m.col(i).data(), m.col(i).data() + m.rows());
		thrust::sort_by_key(d_K.begin(), d_K.end(), d_V.begin(), thrust::greater<float>());
		if (i!=col)
		{
			thrust::copy(d_V.begin(), d_V.end(), m.col(i).data());
		}
		if (i == m.cols()-1)
		{
			thrust::copy(d_K.begin(), d_K.end(), m.col(col).data());
		}
	}
}




std::set<Eigen::VectorXf, mz_comp> pic_seed(const Eigen::MatrixXf & m, float mz_tol, int num_seed)
{
	auto comp = [](const Eigen::VectorXf& lhs, const Eigen::VectorXf& rhs) -> bool {
		return lhs[1] < rhs[1]; 
	};
	std::set<Eigen::VectorXf, mz_comp> ret(comp);

	for (int i =0; i< m.rows(); i++)
	{
		auto it = ret.lower_bound(m.row(i));
		if (ret.size()==0)
		{
			ret.insert(m.row(i));
		}
		else
		{
			if (it == ret.end())
			{
				if (m.row(i)[1] - (*std::prev(ret.end()))[1] > mz_tol)
				{
					ret.insert(m.row(i));
				}
			}
			else if (it == ret.begin())
			{
				if ((*ret.begin())[1] - m.row(i)[1] > mz_tol)
				{
					ret.insert(m.row(i));
				}
			}
			else
			{
				if (  ((*it)[1] - m.row(i)[1] > mz_tol ) && 
					  ( m.row(i)[1] - (*std::prev(it))[1]> mz_tol))
				{
					ret.insert(m.row(i));
				}
			}
		}

		if (ret.size()==num_seed)
		{
			break;
		}
	}


	Eigen::MatrixXf seed(ret.size(), 3);
	int i = 0;
	std::for_each(ret.begin(), ret.end(), [&seed, &i](const Eigen::VectorXf & v) {
		seed.row(i) = v;
		i++; });

	cout << seed << endl;

	return ret;
}

void processLCMS(LCMS & lcms)
{
	hipFree(0);
	cout << "using lcms object in CUDA, and its scan size is: " << lcms.m_massScans.size() << endl;

	Eigen::MatrixXf rmv = lcms.getAll();

	gtic();
	sort_by_col(rmv, 2);
	gtoc();

	gtic();
	pic_seed(rmv, 0.05, 50);
	gtoc();

	//cout << rmv.topRows(10) << endl;
	cout << "##########################"<<endl;
}

