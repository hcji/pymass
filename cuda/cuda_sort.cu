#include "hip/hip_runtime.h"
#include <iostream>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/sort.h>
#include <thrust/generate.h>
#include <thrust/copy.h>
#include <algorithm>
#include <cstdlib>
#include "cuda_sort.h"


using namespace std;



namespace Kernel
{

	static void HandleError(hipError_t err, const char *file, int line)
	{
		if (err != hipSuccess)
		{
			printf("%s in %s at line %d\n", hipGetErrorString(err), file, line);
			exit(EXIT_FAILURE);
		}
	}

    #define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))

	__global__ void find_pics_k(Eigen::Vector3f * seeds_dev, Eigen::Vector3f * regions_dev, int *ids_dev, int n)
	{
		int idx = blockIdx.x * blockDim.x + threadIdx.x;
		if (idx < n)
		{
			Eigen::Vector3f seed = seeds_dev[idx];
			Eigen::Vector3f region = regions_dev[ids_dev[idx]];

		}
		return;
	}
	
	double find_pics(const std::vector<Eigen::Vector3f> & seeds, const std::vector<Eigen::Vector3f> & regions, const std::vector<int> & ids)
	{
		int sz_seed = seeds.size();
		int sz_rg   = regions.size();
		Eigen::Vector3f *seeds_dev;
		HANDLE_ERROR(hipMalloc((void **)&seeds_dev, sizeof(Eigen::Vector3f)*sz_seed));
		HANDLE_ERROR(hipMemcpy(seeds_dev, seeds.data(), sizeof(Eigen::Vector3f)*sz_seed, hipMemcpyHostToDevice));


		Eigen::Vector3f * regions_dev;
		HANDLE_ERROR(hipMalloc((void **)&regions_dev, sizeof(Eigen::Vector3f)*sz_rg));
		HANDLE_ERROR(hipMemcpy(regions_dev, regions.data(), sizeof(Eigen::Vector3f)*sz_rg, hipMemcpyHostToDevice));

		int * ids_dev;
		HANDLE_ERROR(hipMalloc((void **)&ids_dev, sizeof(int)*(sz_seed+1)));
		HANDLE_ERROR(hipMemcpy(ids_dev, ids.data(), sizeof(int)*(sz_seed + 1), hipMemcpyHostToDevice));


		find_pics_k << <(sz_seed + 1023) / 1024, 1024 >> >(seeds_dev, regions_dev, ids_dev, sz_seed);

		hipFree(ids_dev);
		hipFree(regions_dev);
		hipFree(seeds_dev);

		return 0.0;
	}
}



std::stack<clock_t> gtictoc_stack;
void gtic() {
	gtictoc_stack.push(clock());
}

void gtoc() {
	std::cout << "Time elapsed: "
		<< ((double)(clock() - gtictoc_stack.top())) / CLOCKS_PER_SEC
		<< std::endl;
	gtictoc_stack.pop();
}

void printVV(const std::vector<Eigen::Vector3f> & vec, int n)
{
	Eigen::MatrixXf m(vec.size(), 3);
	int i = 0;
	std::for_each(vec.begin(), vec.end(), [&m, &i](const Eigen::Vector3f & v) {
		m.row(i) = v;
		i++; });
	if (n<= vec.size() && n>0)
	{
		cout << m.topRows(n) << endl;
	}
	else
	{
		cout << m << endl;
	}
}

void sort_by_col(Eigen::MatrixXf & m, int col)
{
	for (int i = 0; i < m.cols(); i++)
	{
		thrust::device_vector<float> d_K(m.col(col).data(), m.col(col).data() + m.rows());
		thrust::device_vector<float> d_V(m.col(i).data(), m.col(i).data() + m.rows());
		thrust::sort_by_key(d_K.begin(), d_K.end(), d_V.begin(), thrust::greater<float>());
		if (i!=col)
		{
			thrust::copy(d_V.begin(), d_V.end(), m.col(i).data());
		}
		if (i == m.cols()-1)
		{
			thrust::copy(d_K.begin(), d_K.end(), m.col(col).data());
		}
	}
}

#include <Eigen/Core>
#include <algorithm>
#include <vector>

template <typename Scalar, int rows, int cols, int options, int maxRows, int maxCols>
Eigen::Matrix<Scalar, rows, cols, options, maxRows, maxCols> sortMatrix(
	Eigen::Matrix<Scalar, rows, cols, options, maxRows, maxCols> target, int col
){
	std::vector<Eigen::Matrix<Scalar, 1, cols>> matrixRows;
	for (unsigned int i = 0; i < target.rows(); i++)
		matrixRows.push_back(target.row(i));
	std::sort(
		matrixRows.begin(),
		matrixRows.end(),
		[&col](const Eigen::Matrix<Scalar, 1, cols> & a,const Eigen::Matrix<Scalar, 1, cols> & b)->bool
		{
			return a(0, col) < b(0, col);
		}
	);

	Eigen::Matrix<Scalar, rows, cols, options, maxRows, maxCols> sorted;
	for (unsigned int i = 0; i < matrixRows.size(); i++)
		sorted.row(i) = matrixRows[i];
	return sorted;
}



std::vector<Eigen::Vector3f> pic_seeds(const Eigen::MatrixXf & m, float mz_tol, int num_seed)
{
	auto comp = [](const Eigen::VectorXf& lhs, const Eigen::VectorXf& rhs) -> bool {
		return lhs[1] < rhs[1]; 
	};
	std::set<Eigen::VectorXf, mz_comp> seed_set(comp);

	for (int i =0; i< m.rows(); i++)
	{
		auto it = seed_set.lower_bound(m.row(i));
		if (seed_set.size()==0)
		{
			seed_set.insert(m.row(i));
		}
		else
		{
			if (it == seed_set.end())
			{
				if (m.row(i)[1] - (*std::prev(seed_set.end()))[1] > mz_tol)
				{
					seed_set.insert(m.row(i));
				}
			}
			else if (it == seed_set.begin())
			{
				if ((*seed_set.begin())[1] - m.row(i)[1] > mz_tol)
				{
					seed_set.insert(m.row(i));
				}
			}
			else
			{
				if (  ((*it)[1] - m.row(i)[1] > mz_tol ) && 
					  ( m.row(i)[1] - (*std::prev(it))[1]> mz_tol))
				{
					seed_set.insert(m.row(i));
				}
			}
		}

		if (seed_set.size()==num_seed)
		{
			break;
		}
	}

	std::vector<Eigen::Vector3f> ret(seed_set.size());

	int i = 0;
	std::for_each(seed_set.begin(), seed_set.end(), [&ret, &i](const Eigen::VectorXf & v) {
		ret[i] = v;
		i++; });

	return ret;
}

std::tuple<std::vector<Eigen::Vector3f>, std::vector<int> > regions_of_seeds(LCMS & lcms, const std::vector<Eigen::Vector3f> & seeds, float peak_width, float mz_tol)
{
	std::vector<std::vector<Eigen::Vector3f>> region_vec;
	std::vector<int>                          ids(seeds.size()+1);

	int rows = 0;
	ids[0] = 0;
	for (int i=0; i<seeds.size(); i++)
	{
		Eigen::Vector3f seed = seeds[i];
		std::vector<Eigen::Vector3f> region = lcms.getRegion(seed[0] - peak_width, seed[0] + peak_width, seed[1] - mz_tol, seed[1] + mz_tol);
		region_vec.push_back(region);
		rows += region.size();
		ids[i + 1] = rows;
	}
	std::vector<Eigen::Vector3f>              regions(rows);

	for (int i = 0; i < seeds.size(); i++)
	{
		int sz = ids[i + 1] - ids[i];
		std::copy_n(region_vec[i].begin(), sz, &regions[ids[i]]);
	}

	return std::make_tuple(regions, ids);
}

void processLCMS(LCMS & lcms)
{
	hipFree(0);
	cout << "using lcms object in NVCC, and its scan size is: " << lcms.m_massScans.size() << endl;

	Eigen::MatrixXf rmv = lcms.getAll();

	gtic();
	sort_by_col(rmv, 2);
	gtoc();

	gtic();
	std::vector<Eigen::Vector3f> seeds = pic_seeds(rmv, 0.05f, 8000);
	gtoc();

	gtic();
	std::vector<Eigen::Vector3f> regions;
	std::vector<int>             ids;
	std::tie(regions, ids) = regions_of_seeds(lcms, seeds, 50.0f, 0.05f);
	gtoc();

	gtic();
	double x = Kernel::find_pics(seeds, regions, ids);
	gtoc();
	cout << "Calculated by CUDA kernel: " << x << endl;

	cout << "##########################"<<endl;
}