#include <iostream>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/sort.h>
#include <thrust/generate.h>
#include <thrust/copy.h>
#include <algorithm>
#include <cstdlib>
#include "cuda_sort.h"


using namespace std;

int main() {
	hipFree(0);
	int size = 100*1024*1024;
	thrust::host_vector<float> h_values(size);
	thrust::host_vector<int> h_keys(size);
	
	std::generate(h_values.begin(), h_values.end(), rand);

	return 0;
}