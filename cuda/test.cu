#include <iostream>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/sort.h>
#include <thrust/generate.h>
#include <thrust/copy.h>
#include <algorithm>
#include <cstdlib>
#include "cuda_sort.h"


using namespace std;

int main() {
	hipFree(0);
	int size = 100*1024*1024;
	thrust::host_vector<float> h_values(size);
	thrust::host_vector<int> h_keys(size);
	
	std::generate(h_values.begin(), h_values.end(), rand);

	thrustSort(h_values.data(), h_keys.data(), h_values.size());

	if (size < 100)
	{
		for (int i = 0; i < size; i++)
		{
			cout << "(" << h_values[i] << ", " << h_keys[i] << ")";
		}
	}

	return 0;
}